#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "main.h"

using namespace std;


struct testit {

__device__
  thrust::device_vector<double> operator()(thrust::device_vector<double> plots){

    return plots;
  }
};


/**
 * Host main routine
 */
int main(void){
    printf("Trying to call experiment from main\n");
    experiment();

    printf("Done\n");
    return 0;
}

/*
   Manage the process of loading data, running the experiment
   and saving the results.
*/
__host__
void experiment(){
  for(int k = 0; k < NUMBER_OF_INPUT_FILES; k++){
    std::ostringstream ostr; //output string stream
    ostr << BASE_FILE_PATH << BASE_FILE_NAME << k << FILE_EXT;
    std::fstream myfile(ostr.str().c_str(), std::ios_base::in);
    thrust::host_vector< thrust::host_vector<double> > plots = loadPlots(myfile);
    thrust::host_vector< thrust::host_vector<double> > landUses = loadLandUses(myfile);
    thrust::host_vector< pair<int, int> > assignments = loadAssignments(myfile, plots.size());
    myfile.close();
    cout << "Plots: " << plots.size() << "\tLand Uses: " << landUses.size() << "\tAssignments: " << assignments.size() << endl;
    //vector< pair<int, int> > a;
    //for(int i = 1; i <= ITERATIONS_PER_FILE; i++){
    //  a = assignments;
      //sa(k, i, a);
    //}

    //thrust::device_vector< thrust::device_vector<long double> > d_plots = plots;
    //thrust::transform(d_plots.begin(), d_plots.end(), d_plots.begin(), testit());
  }
}
